
#include <cstdio>
#include <cstdlib>
#include <cstring>
#include <stdexcept>
#include <vector>

#include <hip/hip_runtime.h>

#include "utils.h"
#include "Sebas.h"

using data_type = float;

int main(int argc, char *argv[])
{
    Sebas<data_type> sebas;

    hipStream_t stream = sebas.getStream();
    // hiprandGenerator_t gen = NULL;
    // hiprandOrdering_t order = HIPRAND_ORDERING_PSEUDO_BEST;

    data_type *d_data = nullptr;

    const int n = 30;

    const data_type mean = 1.0f;
    const data_type stddev = 2.0f;

    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_data), sizeof(data_type) * n));

    // /* Generate n floats on device */
    sebas.normalDistribution(d_data, n, mean, stddev);
    std::vector<data_type> h_data(n, 0);
    // /* Copy data to host */
    CUDA_CHECK(hipMemcpyAsync(h_data.data(), d_data,
                               sizeof(data_type) * h_data.size(),
                               hipMemcpyDeviceToHost, stream));

    // /* Sync stream */
    CUDA_CHECK(hipStreamSynchronize(stream));

 
    printf("normal\n");
    print_vector(h_data);
    printf("=====\n");

    const  int low = 10;
    const int high = 20;

    unsigned int *d_data2 = nullptr;
    CUDA_CHECK(hipMalloc(reinterpret_cast<void **>(&d_data2), sizeof(unsigned int) * n));

    /* Generate n unsigned ints on device */
    sebas.uniformIntDistribution(d_data2, n, low, high);
    std::vector<unsigned int> h_data2(n, 0);
    /* Copy data to host */
    CUDA_CHECK(hipMemcpyAsync(h_data2.data(), d_data2,
                               sizeof(unsigned int) * h_data2.size(),
                               hipMemcpyDeviceToHost, stream));
    
    /* Sync stream */
    CUDA_CHECK(hipStreamSynchronize(stream));

    printf("uniform int\n");
    print_vector(h_data2);
    printf("=====\n");





 

    return EXIT_SUCCESS;
}