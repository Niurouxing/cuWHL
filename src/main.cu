
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "BsP.h"

void printMatrix(float *matrix, int rows, int cols)
{
    float *host = new float[rows * cols];
    hipMemcpy(host, matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%f ", host[i + j * rows]);
        }
        printf("\n");
    }

    delete[] host;
    fflush(stdout);
}

void printMatrix(int *matrix, int rows, int cols)
{
    int *host = new int[rows * cols];
    hipMemcpy(host, matrix, rows * cols * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d ", host[i + j * rows]);
        }
        printf("\n");
    }

    delete[] host;
    fflush(stdout);
}

void printVector(float *vector, int size)
{
    float *host = new float[size];
    hipMemcpy(host, vector, size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++)
    {
        printf("%f ", host[i]);
    }
    printf("\n");

    delete[] host;
    fflush(stdout);
}

void printVector(int *vector, int size)
{
    int *host = new int[size];
    hipMemcpy(host, vector, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++)
    {
        printf("%d ", host[i]);
    }
    printf("\n");

    delete[] host;
    fflush(stdout);
}

void printCube(float *cube, int rows, int cols, int pages)
{
    //  print for each page
    for (int i = 0; i < pages; i++)
    {
        printf("Page %d\n", i);
        printMatrix(cube + i * rows * cols, rows, cols);
    }
}

void printCube(int *cube, int rows, int cols, int pages)
{
    //  print for each page
    for (int i = 0; i < pages; i++)
    {
        printf("Page %d\n", i);
        printMatrix(cube + i * rows * cols, rows, cols);
    }
}

int main()
{
    constexpr int Tx = 32;
    constexpr int Rx = 32;

    constexpr int dm = 4;

    auto det = Detection<Tx, Rx, QAM<16, RD>>();

    det.generate();

    hipDeviceSynchronize();

    // printf("H\n");
    // printMatrix(det.H, 2 * Rx, 2 * Tx);


    // printf("Rx\n");
    // printVector(det.RxSymbols, 2 * Rx);



    auto bsp = BsP<Tx, Rx, QAM<16, RD>, dm>();

    bsp.execute(det);

    hipDeviceSynchronize();



    // printf("gamma\n");
    // printMatrix(bsp.gamma, det.ConSize, 2 * Tx);




    // printf("sIndex\n");
    // printCube(bsp.sIndex, dm, 2 * Tx, 2 * Rx);

    // // beta
    // printf("beta\n");
    // printCube(bsp.beta, det.ConSize, 2 * Rx, 2 * Tx);

    // printf("gamma\n");
    // printMatrix(bsp.gamma, det.ConSize, 2 * Tx);


    // printf("alpha\n");
    // // [ConSize][2 * TxAntNum][2 * RxAntNum]
    // printCube(bsp.alpha, det.ConSize, 2 * Tx, 2 * Rx);

    // printf("Px\n");
    // printCube(bsp.Px, det.ConSize, 2 * Tx, 2 * Rx);

    printf("HtH\n");
    printMatrix(bsp.HtH, 2 * Tx, 2 * Tx);

    printf("Tx\n");
    printVector(det.TxSymbols, 2 * Tx);
    printVector(det.TxIndices, 2 * Tx);
    printf("Est\n");
    printVector(bsp.HtY, 2 * Tx);

    printf("TxEst\n");
    printVector(bsp.TxEst, 2 * Tx);


}

