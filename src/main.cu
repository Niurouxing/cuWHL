
#include <stdio.h>
#include "hip/hip_runtime.h"
#include "BsP.h"

void printMatrix(float *matrix, int rows, int cols)
{
    float *host = new float[rows * cols];
    hipMemcpy(host, matrix, rows * cols * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%f ", host[i + j * rows]);
        }
        printf("\n");
    }

    delete[] host;
}

void printMatrix(int *matrix, int rows, int cols)
{
    int *host = new int[rows * cols];
    hipMemcpy(host, matrix, rows * cols * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < rows; i++)
    {
        for (int j = 0; j < cols; j++)
        {
            printf("%d ", host[i + j * rows]);
        }
        printf("\n");
    }

    delete[] host;
}

void printVector(float *vector, int size)
{
    float *host = new float[size];
    hipMemcpy(host, vector, size * sizeof(float), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++)
    {
        printf("%f ", host[i]);
    }
    printf("\n");

    delete[] host;
}

void printVector(int *vector, int size)
{
    int *host = new int[size];
    hipMemcpy(host, vector, size * sizeof(int), hipMemcpyDeviceToHost);

    for (int i = 0; i < size; i++)
    {
        printf("%d ", host[i]);
    }
    printf("\n");

    delete[] host;
}

void printCube(float *cube, int rows, int cols, int pages)
{
    //  print for each page
    for (int i = 0; i < pages; i++)
    {
        printf("Page %d\n", i);
        printMatrix(cube + i * rows * cols, rows, cols);
    }
}

void printCube(int *cube, int rows, int cols, int pages)
{
    //  print for each page
    for (int i = 0; i < pages; i++)
    {
        printf("Page %d\n", i);
        printMatrix(cube + i * rows * cols, rows, cols);
    }
}

int main()
{
    constexpr int Tx = 3;
    constexpr int Rx = 4;

    constexpr int dm = 3;

    auto det = Detection<Tx, Rx>();

    det.generate();

    auto bsp = BsP<Tx, Rx, QAM<16,RD>,dm>();

    bsp.execute(det);

    printf("H\n");
    printMatrix(bsp.HtH, 2 * Tx, 2 * Tx);

    printf("alpha\n");
    // [ConSize][2 * TxAntNum][2 * RxAntNum]
    printCube(bsp.alpha, det.ConSize, 2 * Tx, 2 * Rx);

    printf("gamma\n");
    printMatrix(bsp.gamma, det.ConSize, 2 * Tx);

    printf("Tx\n");
    printVector(det.TxSymbols, 2 * Tx);
    printVector(det.TxIndices, 2 * Tx);

    printf("Est\n");
    printVector(bsp.HtY, 2 * Tx);

    printf("sIndex\n");
    printCube(bsp.sIndex,dm, 2 * Tx, 2 * Rx);
}